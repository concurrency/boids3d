#include "hip/hip_runtime.h"
/*
 *	occoids_cuda.cu -- boid GPU code
 *	Copyright (C) 2012,2013,2017  Fred Barnes <F.R.M.Barnes@gmail.com>
 *
 *	This program is free software: you can redistribute it and/or modify
 *	it under the terms of the GNU General Public License as published by
 *	the Free Software Foundation, either version 3 of the License, or
 *	(at your option) any later version.
 *
 *	This program is distributed in the hope that it will be useful,
 *	but WITHOUT ANY WARRANTY; without even the implied warranty of
 *	MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *	GNU General Public License for more details.
 *
 *	You should have received a copy of the GNU General Public License
 *	along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */


#include <stdio.h>

#include <hip/hip_runtime.h>
#include <math_functions.h>
#include <hip/hip_math_constants.h>

#include <dmem_if.h>
#include <ocuda_ctypes.h>

#include "occoids_ctypes.h"


/*{{{  dev_jobinfo_t: instance parameter (per chunk of work)*/

/* (this avoids trying to pass too many parameters to the GPU kernel) */

typedef struct TAG_dev_jobinfo {
	int astart, acount;		/* start and count in the global agent array */
	int act_cycle;			/* activity cycle (0 or 1) */
} dev_jobinfo_t;

/*}}}*/
/*{{{  constjobinfo_t: constant information for a series of runs*/

typedef struct TAG_constjobinfo {
	genparms_t *host_parms;
	int parms_size;

	agent_info_t *host_all_agents[2];
	int all_agents_size;

	int *host_viewable;
	int viewable_size;

	int *host_viewcount;
	int viewcount_size;

	int *host_viewmap;
	int viewmap_size;
} constjobinfo_t;

/*}}}*/
/*{{{  dev_constjobinfo_t: constant job information pointers*/

typedef struct TAG_dev_constjobinfo {
	genparms_t *dev_parms;			/* simulation parameters */
	agent_info_t *dev_all_agents0;		/* agent data */
	agent_info_t *dev_all_agents1;		/* agent data */
	int *dev_viewable;			/* viewable information */
	int *dev_viewcount;			/* how many in each location's viewer */
	int *dev_viewmap;			/* view-map (how the world is connected) */
} dev_constjobinfo_t;

/*}}}*/

__device__ inline float angle_diff (const float a, const float b) /*{{{*/
{
	float r = a - b;

	if (r < -HIP_PI_F) {
		r += (2.0 * HIP_PI_F);
	} else if (r > HIP_PI_F) {
		r -= (2.0 * HIP_PI_F);
	}
	r = fabsf (r);
	return r;
}
/*}}}*/
__device__ inline float vec_mag2 (vector_t vec) /*{{{*/
{
	return (vec.x * vec.x) + (vec.y * vec.y) + (vec.z * vec.z);
}
/*}}}*/
__device__ inline void boid_combined_rules (const genparms_t *devparms, int my_id, agent_info_t *result, agent_info_t *all_agents, int nagents, int *viewable, int nview, int *viewcount, int nvcount, int *viewmap, int vmstride) /*{{{*/
{
	float my_angle_a = atan2f (all_agents[my_id].vel.x, all_agents[my_id].vel.y);
	float my_angle_b = atan2f (all_agents[my_id].vel.x, all_agents[my_id].vel.z);
	float my_vel2 = vec_mag2 (all_agents[my_id].vel);
	bool not_moving = (my_vel2 < devparms->epsilon);

	int nboids = 0, nobst = 0;

	vector_t com = {0.0, 0.0, 0.0, 0.0};		// centre of mass
	vector_t rpush = {0.0, 0.0, 0.0, 0.0};		// repulsion push
	vector_t opush = {0.0, 0.0, 0.0, 0.0};		// obstacle push
	vector_t perc_vel = {0.0, 0.0, 0.0, 0.0};	// perceived velocity

	vector_t accel = {0.0, 0.0, 0.0, 0.0};

	int i, v;
	int cell = all_agents[my_id].locn & 0x000fffff;
	int boundary = (all_agents[my_id].locn >> 26) & 0x3f;

	result->type_id = all_agents[my_id].type_id;
	result->locn = all_agents[my_id].locn;
	result->vel = all_agents[my_id].vel;
	result->radius = all_agents[my_id].radius;
	result->colour = all_agents[my_id].colour;

	/* for each cell in the viewable space */
	for (v = 0; v <= vmstride; v++) {
		int cidx, cvcount, cvstart;

		if (v < vmstride) {
			cidx = viewmap[(cell * vmstride) + v];
		} else {
			cidx = cell;
		}

		cvcount = viewcount[cidx];
		cvstart = (cidx * devparms->vperboid);

		/* for each thing in the viewable cell */
		for (i=0; i<cvcount; i++) {
			int a_id = viewable[cvstart + i];

			if (a_id == my_id) {
				/* can't see ourselves */
			} else {
				vector_t rel_pos;
				bool visible;
				int atype = (all_agents[a_id].type_id >> 24) & 0xff;

				/*{{{  calculate relative position*/
				rel_pos.x = all_agents[a_id].pos.x - all_agents[my_id].pos.x;
				rel_pos.y = all_agents[a_id].pos.y - all_agents[my_id].pos.y;
				rel_pos.z = all_agents[a_id].pos.z - all_agents[my_id].pos.z;

				if ((boundary & 0x05) == 0) {
					/* not at left/right edge */
				} else if (rel_pos.x > 2.0) {
					/* can't possibly be this much */
					rel_pos.x -= (float)devparms->width;
				} else if (rel_pos.x < -2.0) {
					rel_pos.x += (float)devparms->width;
				}
				if ((boundary & 0x0a) == 0) {
					/* not at top/bottom edge */
				} else if (rel_pos.y > 2.0) {
					/* can't possibly be this much */
					rel_pos.y -= (float)devparms->height;
				} else if (rel_pos.y < -2.0) {
					rel_pos.y += (float)devparms->height;
				}
				if ((boundary & 0x30) == 0) {
					/* not at front/back edge */
				} else if (rel_pos.z > 2.0) {
					/* can't possibly be this much */
					rel_pos.z -= (float)devparms->depth;
				} else if (rel_pos.z < -2.0) {
					rel_pos.z += (float)devparms->depth;
				}
				/*}}}*/
				/*{{{  is it visible?*/
				if (vec_mag2 (rel_pos) > devparms->vrad_sq) {
					visible = false;		/* too far away */
				} else if (atype == 2) {
					visible = true;			/* obstacles always visible */
				} else if (not_moving) {
					visible = true;			/* not moving, look around */
				} else if (angle_diff (atan2f (rel_pos.x, rel_pos.y), my_angle_a) > devparms->vmaxdiff) {
					visible = false;		/* out of field-of-view */
				} else if (angle_diff (atan2f (rel_pos.x, rel_pos.z), my_angle_b) > devparms->vmaxdiff) {
					visible = false;		/* out of field-of-view */
				} else {
					visible = true;
				}
				/*}}}*/

				if (visible && (atype == 1)) {
					/*{{{  boid*/
					nboids++;
					/* centre of mass (1) */
					com.x += rel_pos.x;
					com.y += rel_pos.y;
					com.z += rel_pos.z;
					/* repulsion (1) */
					if (vec_mag2 (rel_pos) < devparms->rdist_sq) {
						rpush.x -= rel_pos.x;
						rpush.y -= rel_pos.y;
						rpush.z -= rel_pos.z;
					}
					/* mean velocity (1) */
					perc_vel.x += all_agents[a_id].vel.x;
					perc_vel.y += all_agents[a_id].vel.y;
					perc_vel.z += all_agents[a_id].vel.z;
					/*}}}*/
				} else if (visible && (atype == 2)) {
					/*{{{  obstacle*/
					float dist;

					nobst++;
					/* obstacle rule (1) */
					dist = sqrtf (vec_mag2 (rel_pos)) - all_agents[a_id].radius;

					if (dist < 0.0) {
						/* past soft threshold, push back hard */
						opush.x -= rel_pos.x;
						opush.y -= rel_pos.y;
						opush.z -= rel_pos.z;
					} else if (dist < devparms->sthres) {
						/* inside soft threshold, push back a variable amount */
						float scl = (1.0 - (dist / devparms->sthres));

						opush.x -= (rel_pos.x * scl);
						opush.y -= (rel_pos.y * scl);
						opush.z -= (rel_pos.z * scl);
					}
					/*}}}*/
				}
			}
		}
	}

	/*{{{  centre of mass (2)*/
	if (nboids > 0) {
		com.x /= (float)nboids;
		com.y /= (float)nboids;
		com.z /= (float)nboids;

		accel.x += (com.x / devparms->cmfrac);
		accel.y += (com.y / devparms->cmfrac);
		accel.z += (com.z / devparms->cmfrac);
	}

	/*}}}*/
	/*{{{  repulsion (2)*/
	accel.x += (rpush.x / devparms->rfrac);
	accel.y += (rpush.y / devparms->rfrac);
	accel.z += (rpush.z / devparms->rfrac);

	/*}}}*/
	/*{{{  mean velocity (2)*/
	if (nboids > 0) {
		perc_vel.x /= (float)nboids;
		perc_vel.y /= (float)nboids;
		perc_vel.z /= (float)nboids;
	}

	perc_vel.x -= result->vel.x;
	perc_vel.y -= result->vel.y;
	perc_vel.z -= result->vel.z;
	accel.x += (perc_vel.x / devparms->mvfrac);
	accel.y += (perc_vel.y / devparms->mvfrac);
	accel.z += (perc_vel.z / devparms->mvfrac);

	/*}}}*/
	/*{{{  obstacle rule (2)*/
	if (nobst > 0) {
		accel.x += (opush.x / devparms->ofrac);
		accel.y += (opush.y / devparms->ofrac);
		accel.z += (opush.z / devparms->ofrac);
	}

	/*}}}*/
	/*{{{  accelerate!*/
	result->vel.x += (accel.x / devparms->saccel);
	result->vel.y += (accel.y / devparms->saccel);
	result->vel.z += (accel.z / devparms->saccel);

	if (fabsf (result->vel.x) < devparms->epsilon) {
		result->vel.x = 0.0;
	}
	if (fabsf (result->vel.y) < devparms->epsilon) {
		result->vel.y = 0.0;
	}
	if (fabsf (result->vel.z) < devparms->epsilon) {
		result->vel.z = 0.0;
	}

	my_vel2 = vec_mag2 (result->vel);
	if (my_vel2 > devparms->slimit_sq) {
		float tmag = my_vel2 / devparms->slimit_sq;

		result->vel.x = result->vel.x / tmag;
		result->vel.y = result->vel.y / tmag;
		result->vel.z = result->vel.z / tmag;
	}

	/*}}}*/
	/*{{{  compute new position and movement info*/
	result->pos.x = all_agents[my_id].pos.x + result->vel.x;
	result->pos.y = all_agents[my_id].pos.y + result->vel.y;
	result->pos.z = all_agents[my_id].pos.z + result->vel.z;

#if 0
	/* round new position into grid */
	if (result->pos.x < 0.0) {
		result->pos.x += (float)devparms->width;
	} else if (result->pos.x >= (float)devparms->width) {
		result->pos.x -= (float)devparms->width;
	}
	if (result->pos.y < 0.0) {
		result->pos.y += (float)devparms->height;
	} else if (result->pos.y >= (float)devparms->height) {
		result->pos.y -= (float)devparms->height;
	}

	/* compute delta -- note: always positive if wrapping */
	int xdelta = (int)(truncf (result->pos.x) - truncf (all_agents[my_id].pos.x));
	int ydelta = (int)(truncf (result->pos.y) - truncf (all_agents[my_id].pos.y));

	if ((xdelta == -1) || (xdelta > 2)) {
		/* moving left */
		result->locn |= 0x01000000;
	} else if ((xdelta == 1) || (xdelta < -2)) {
		/* moving right */
		result->locn |= 0x04000000;
	}
	if ((ydelta == -1) || (ydelta > 2)) {
		/* moving up */
		result->locn |= 0x02000000;
	} else if ((ydelta == 1) || (ydelta < -2)) {
		/* moving down */
		result->locn |= 0x08000000;
	}
#endif

	/*}}}*/
}
/*}}}*/
__global__ void boid_gpu5_kernel2 (dev_constjobinfo_t cinfo, dev_jobinfo_t jinfo) /*{{{*/
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int my_id;
	agent_info_t *arry, *rarry;

	if (i >= jinfo.acount) {
		return;				/* out of range */
	}

	my_id = jinfo.astart + i;
	if (jinfo.act_cycle == 0) {
		arry = cinfo.dev_all_agents0;
		rarry = cinfo.dev_all_agents1;
	} else {
		arry = cinfo.dev_all_agents1;
		rarry = cinfo.dev_all_agents0;
	}

	if (((arry[my_id].type_id >> 24) & 0xff) == 1) {
		boid_combined_rules (cinfo.dev_parms, my_id, &(rarry[my_id]),
					arry, cinfo.dev_parms->maxagents,
					cinfo.dev_viewable, cinfo.dev_parms->viewsize,
					cinfo.dev_viewcount, cinfo.dev_parms->vperboid,
					cinfo.dev_viewmap, cinfo.dev_parms->ndir);
	}
}
/*}}}*/

int __get_last_cuda_error (const char *msg, const char *file, const int line) /*{{{*/
{
	hipError_t err = hipGetLastError ();

	if (err != hipSuccess) {
		fprintf (stderr, "%s(%d): CUDA error: %s: (%d): %s\n", file, line, msg, (int)err, hipGetErrorString (err));
		return 1;
	}
	return 0;
}
/*}}}*/

#define get_last_cuda_error(MSG) __get_last_cuda_error (MSG, __FILE__, __LINE__)

static inline void real_cuda_alloc_global (ocuda_devinfo_t *device, int all_agents_addr_0, int all_agents_addr_1, int viewable_addr, int viewcount_addr, int viewmap_addr, int parms_addr, int *cpuptr, int *gpuptr) /*{{{*/
{
	constjobinfo_t *hjob;
	dev_constjobinfo_t *djob;

	hipSetDevice (device->dnum);

	hjob = (constjobinfo_t *)malloc (sizeof (constjobinfo_t));
	if (!hjob) {
		fprintf (stderr, "real_cuda_alloc_global(): malloc() failed!\n");
		*cpuptr = 0;
		*gpuptr = 0;
		return;
	}

	hjob->host_parms = (genparms_t *)parms_addr;
	hjob->parms_size = sizeof (genparms_t);
	hjob->host_all_agents[0] = (agent_info_t *)all_agents_addr_0;
	hjob->host_all_agents[1] = (agent_info_t *)all_agents_addr_1;
	hjob->all_agents_size = (sizeof (agent_info_t) * hjob->host_parms->maxagents);
	hjob->host_viewable = (int *)viewable_addr;
	hjob->viewable_size = (sizeof (int) * hjob->host_parms->viewsize);
	hjob->host_viewcount = (int *)viewcount_addr;
	hjob->viewcount_size = (sizeof (int) * hjob->host_parms->nloc);
	hjob->host_viewmap = (int *)viewmap_addr;
	hjob->viewmap_size = (sizeof (int)  * hjob->host_parms->nloc * hjob->host_parms->ndir);

	*cpuptr = (int)hjob;

	djob = (dev_constjobinfo_t *)malloc (sizeof (dev_constjobinfo_t));
	if (!djob) {
		fprintf (stderr, "real_cuda_alloc_global(): malloc() failed!\n");
		*gpuptr = 0;
		return;
	}

	hipMalloc ((void **)&(djob->dev_parms), hjob->parms_size);
	hipMalloc ((void **)&(djob->dev_all_agents0), hjob->all_agents_size);
	hipMalloc ((void **)&(djob->dev_all_agents1), hjob->all_agents_size);
	hipMalloc ((void **)&(djob->dev_viewable), hjob->viewable_size);
	hipMalloc ((void **)&(djob->dev_viewcount), hjob->viewcount_size);
	hipMalloc ((void **)&(djob->dev_viewmap), hjob->viewmap_size);

	if (get_last_cuda_error ("malloc")) {
		*gpuptr = 0;
		free (djob);
		return;
	}

	*gpuptr = (int)djob;

#if 0
fprintf (stderr, "real_cuda_alloc_global(): dev_all_agents0 = %p, dev_all_agents1 = %p, host_all_agents[0] = %p, host_all_agents[1] = %p, maxagents = %d, all_agents_size = %d\n",
		djob->dev_all_agents0, djob->dev_all_agents1, hjob->host_all_agents[0], hjob->host_all_agents[1], hjob->host_parms->maxagents, hjob->all_agents_size);
#endif

	return;
}
/*}}}*/
static inline void real_cuda_free_global (ocuda_devinfo_t *device, int *cpuptr, int *gpuptr) /*{{{*/
{
	constjobinfo_t *hjob = (constjobinfo_t *)(*cpuptr);
	dev_constjobinfo_t *djob = (dev_constjobinfo_t *)(*gpuptr);

	hipSetDevice (device->dnum);

	if (djob) {
		if (djob->dev_all_agents0) {
			hipFree (djob->dev_all_agents0);
		}
		if (djob->dev_all_agents1) {
			hipFree (djob->dev_all_agents1);
		}
		if (djob->dev_viewcount) {
			hipFree (djob->dev_viewcount);
		}
		if (djob->dev_viewable) {
			hipFree (djob->dev_viewable);
		}
		if (djob->dev_viewmap) {
			hipFree (djob->dev_viewmap);
		}
		if (djob->dev_parms) {
			hipFree (djob->dev_parms);
		}

		free (djob);
		*gpuptr = 0;
	}
	if (hjob) {
		free (hjob);
		*cpuptr = 0;
	}

	return;
}
/*}}}*/
static inline void real_cuda_copy_viewmap (ocuda_devinfo_t *device, int cpuptr, int gpuptr) /*{{{*/
{
	constjobinfo_t *hjob = (constjobinfo_t *)cpuptr;
	dev_constjobinfo_t *djob = (dev_constjobinfo_t *)gpuptr;

	hipSetDevice (device->dnum);
	get_last_cuda_error ("setdevice");

	hipMemcpy (djob->dev_viewmap, hjob->host_viewmap, hjob->viewmap_size, hipMemcpyHostToDevice);
	get_last_cuda_error ("memcpy");

	return;
}
/*}}}*/
static inline void real_cuda_copy_global (ocuda_devinfo_t *device, int act_cycle, int cpuptr, int gpuptr) /*{{{*/
{
	constjobinfo_t *hjob = (constjobinfo_t *)cpuptr;
	dev_constjobinfo_t *djob = (dev_constjobinfo_t *)gpuptr;

	hipSetDevice (device->dnum);
	get_last_cuda_error ("setdevice");

#if 0
fprintf (stderr, "real_cuda_copy_global(): copying!\n");
#endif
	if (act_cycle == 0) {
		hipMemcpy (djob->dev_all_agents0, hjob->host_all_agents[0], hjob->all_agents_size, hipMemcpyHostToDevice);
	} else {
		hipMemcpy (djob->dev_all_agents1, hjob->host_all_agents[1], hjob->all_agents_size, hipMemcpyHostToDevice);
	}
	hipMemcpy (djob->dev_viewable, hjob->host_viewable, hjob->viewable_size, hipMemcpyHostToDevice);
	hipMemcpy (djob->dev_viewcount, hjob->host_viewcount, hjob->viewcount_size, hipMemcpyHostToDevice);
	hipMemcpy (djob->dev_parms, hjob->host_parms, hjob->parms_size, hipMemcpyHostToDevice);
	get_last_cuda_error ("memcpy");

#if 0
	if (get_last_cuda_error ("memcpy")) {
		fprintf (stderr, "real_cuda_copy_global(): arf: dev_all_agents=%p, dev_viewable=%p, dev_viewcount=%p, dev_parms=%p\n",
				djob->dev_all_agents, djob->dev_viewable, djob->dev_viewcount, djob->dev_parms);
	}
#endif

	return;
}
/*}}}*/
static inline void real_cuda_copyback_global (ocuda_devinfo_t *device, int act_cycle, int start, int count, int cpuptr, int gpuptr) /*{{{*/
{
	constjobinfo_t *hjob = (constjobinfo_t *)cpuptr;
	dev_constjobinfo_t *djob = (dev_constjobinfo_t *)gpuptr;
	int copysize = count * sizeof (agent_info_t);

	hipSetDevice (device->dnum);
	get_last_cuda_error ("setdevice");

	if (act_cycle == 0) {
#if 0
fprintf (stderr, "real_cuda_copyback_global(): dst=%p, src=%p, size=%d\n", &(hjob->host_all_agents[1][start]), &(djob->dev_all_agents1[start]), copysize);
#endif
		hipMemcpy (&(hjob->host_all_agents[1][start]), &(djob->dev_all_agents1[start]), copysize, hipMemcpyDeviceToHost);
	} else {
		hipMemcpy (&(hjob->host_all_agents[0][start]), &(djob->dev_all_agents0[start]), copysize, hipMemcpyDeviceToHost);
	}
	if (get_last_cuda_error ("memcpy")) {
		fprintf (stderr, "act_cycle = %d, dev_all_agents0 = %p, dev_all_agents1 = %p, host_all_agents[0] = %p, host_all_agents[1] = %p, all_agents_size = %d\n",
				act_cycle, djob->dev_all_agents0, djob->dev_all_agents1, hjob->host_all_agents[0],
				hjob->host_all_agents[1], hjob->all_agents_size);
	}
	return;
}
/*}}}*/
static inline void real_cuda_doboids1 (ocuda_devinfo_t *device, int act_cycle, int astart0, int acount0, int gpuptr) /*{{{*/
{
	dev_jobinfo_t djob0;
	dev_constjobinfo_t *cjinfo = (dev_constjobinfo_t *)gpuptr;

	hipSetDevice (device->dnum);

	/* populate for-device structure */

	djob0.astart = astart0;
	djob0.acount = acount0;
	djob0.act_cycle = act_cycle;

	/* go run it! */
	int threadsPerBlock = 256;
	int blocksPerGrid0 = (acount0 + threadsPerBlock - 1) / threadsPerBlock;

#if 0
fprintf (stderr, "real_cuda_doboids1 [%d]: launching kernel (%d,%d) args (->dev_all_agents=%p, ->nagents=%d)\n", device,
			blocksPerGrid0, threadsPerBlock, cjinfo->dev_all_agents, djob0.nagents);
#endif
	boid_gpu5_kernel2 <<< blocksPerGrid0, threadsPerBlock >>> (*cjinfo, djob0);
	// boid_gpu5_kernel2 <<< blocksPerGrid0, threadsPerBlock, 0, devinfo[device].stream0 >>> (*cjinfo, djob0);
	// hipStreamSynchronize (devinfo[device].stream0);
	get_last_cuda_error ("exec");

	/* copy the results back */
	// hipMemcpyAsync (ljob0.host_res_vel, djob0.dev_res_vel, ljob0.res_vel_size, hipMemcpyDeviceToHost, devinfo[device].stream0);
	// hipMemcpy (ljob0.host_res_vel, djob0.dev_res_vel, ljob0.res_vel_size, hipMemcpyDeviceToHost);
	// hipStreamSynchronize (devinfo[device].stream0);
	// get_last_cuda_error ("memcpy");

	// hipFree (djob0.dev_res_vel);

	return;
}
/*}}}*/
static inline void real_cuda_runcycle (ocuda_devinfo_t *device, int act_cycle, int start, int count, int cpuptr, int gpuptr) /*{{{*/
{
	real_cuda_copy_global (device, act_cycle, cpuptr, gpuptr);
	real_cuda_doboids1 (device, act_cycle, start, count, gpuptr);
	real_cuda_copyback_global (device, act_cycle, start, count, cpuptr, gpuptr);
}
/*}}}*/

extern "C" {
	/* PROC C.cuda.alloc.global (OCUDA.DEVINFO device, VAL INT all.agents.addr.0, all.agents.addr.1, viewable.addr, viewcount.addr, viewmap.addr, parms.addr, RESULT CPUPTR cpu, gpu) */
	__host__ void _cuda_alloc_global (int *ws) { real_cuda_alloc_global ((ocuda_devinfo_t *)(ws[0]), (int)(ws[1]), (int)(ws[2]), (int)(ws[3]), (int)(ws[4]), (int)(ws[5]), (int)(ws[6]), (int *)(ws[7]), (int *)(ws[8])); }
	/* PROC C.cuda.free.global (OCUDA.DEVINFO device, CPUPTR cpu, CPUPTR gpu) */
	__host__ void _cuda_free_global (int *ws) { real_cuda_free_global ((ocuda_devinfo_t *)(ws[0]), (int *)(ws[1]), (int *)(ws[2])); }
	/* PROC C.cuda.copy.global (OCUDA.DEVINFO device, VAL CPUPTR cpu, gpu) */
	__host__ void _cuda_copy_viewmap (int *ws) { real_cuda_copy_viewmap ((ocuda_devinfo_t *)(ws[0]), (int)(ws[1]), (int)(ws[2])); }
	/* PROC C.cuda.copy.global (OCUDA.DEVINFO device, VAL INT act.cycle, VAL CPUPTR cpu, gpu) */
	__host__ void _cuda_copy_global (int *ws) { real_cuda_copy_global ((ocuda_devinfo_t *)(ws[0]), (int)(ws[1]), (int)(ws[2]), (int)(ws[3])); }
	/* PROC [BC].cuda.doboids1 (OCUDA.DEVINFO device, VAL INT act.cycle, astart0, acount0, VAL CPUPTR gpu) */
	__host__ void _cuda_doboids1 (int *ws) { real_cuda_doboids1 ((ocuda_devinfo_t *)(ws[0]), (int)(ws[1]), (int)(ws[2]), (int)(ws[3]), (int)(ws[4])); }
	/* PROC c.cuda.copyback.global (OCUDA.DEVINFO device, VAL INT act.cycle, astart, acount, VAL CPUPTR cpu, gpu) */
	__host__ void _cuda_copyback_global (int *ws) { real_cuda_copyback_global ((ocuda_devinfo_t *)(ws[0]), (int)(ws[1]), (int)(ws[2]), (int)(ws[3]), (int)(ws[4]), (int)(ws[5])); }

	/* PROC [BC].cuda.runcycle (OCUDA.DEVINFO device, VAL INT act.cycle, astart0, acount0, VAL CPUPTR cpu, gpu) */
	__host__ void _cuda_runcycle (int *ws) { real_cuda_runcycle ((ocuda_devinfo_t *)(ws[0]), (int)(ws[1]), (int)(ws[2]), (int)(ws[3]), (int)(ws[4]), (int)(ws[5])); }
}

